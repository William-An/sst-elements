
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void helloWorld(int* src, int N) {
    for (int i = 0; i < N; i++) {
        src[i] = i + i;
    }
}

int main() {
    int* src;
    int* src_device;
    int N = 32;

    src = (int*)malloc(N * sizeof(int));
    hipMalloc(&src_device, N*sizeof(int));

    for(int i = 0; i < N; i++) {
        src[i] = i;
    }

    hipMemcpy(src_device, src, N*sizeof(int), hipMemcpyHostToDevice);

        helloWorld<<<1, 1, 1>>>(src_device, N);

    hipMemcpy(src, src_device, N*sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) {
        printf("src[%d] = %d\n", i, src[i]);
    }

    return 0;
}
